
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

/*
__global__ void kernel10(int *a)
{
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x); 
}

*/


__global__ void kernel10(int *a)
{
  printf("Hello from thread %d and %d in block %d and %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);  
}



int main() {


  int n=20;
  int memSize = n*sizeof(int);

  int *a, *d_a;
  a = (int*) malloc (n*sizeof(*a));
  hipMalloc( (void**) &d_a, memSize);


  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);

  dim3 block(4,2); 
  dim3 grid(3,2);
  kernel10<<<grid,block>>>(d_a);

  hipMemcpy( a, d_a, memSize, hipMemcpyDeviceToHost);

  	
  hipFree(d_a);
  free(a);
  
  return 0;

}

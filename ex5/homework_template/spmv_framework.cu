#include "hip/hip_runtime.h"
/* 
*   This provides the framework for an SpMV GPU kernel (double precision).
*
*   The first part of the program is reading in a sparse matrix according to 
*   the example_read.c from Matrix Market: 
*   http://math.nist.gov/MatrixMarket/mmio-c.html
*   
*   In the second part, the vectors are initialized and Matrix and vectors are
*   sent to the GPU.
*   
*   The third part is the actual SpMV kernel. This is where you configure your
*   compute grid and call your kernel.
*
*   In the fourth part you validate your results. How you do this is up to you,
*   but you have to explain and document it.
*   
*   Compile this file with
*   nvcc -I./ mmio.cu spmv_framework.cu -o spmv
*   
*   If you encounter problems: hanzt@icl.utk.edu
*       
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <vector>
#include <utility>  // pair
#include <mmio.h>

/**
    Purpose
    -------
    Returns true if first element of a is less than first element of b.
    Ignores second element. Used for sorting pairs,
    std::pair< int, magmaDoubleComplex >, of column indices and values.
*/
static bool compare_first(
    const std::pair< int, double >& a,
    const std::pair< int, double >& b )
{
    return (a.first < b.first);
}

__global__ void spmv_kernel( int M, int k, int N, int nz, int *drow, int *dcol, double *dval, double *dx, double *dy)
{

    int tid = threadIdx.x, bid = blockIdx.x, bdim = blockDim.x, Mrow, bvec;
    if(bid*bdim + tid < M*k)
    {
        Mrow = (int)((bid*bdim + tid)/k);
        bvec = bid*bdim + tid - Mrow*k;
        for(int i = drow[Mrow]; i < drow[Mrow + 1]; i++)
        {
            dy[bvec*M + Mrow] += dval[i]*dx[bvec*M + dcol[i]];
        }
    }
}

double error_norm(int M, int k, double *y, double *y2)
{
    double temp, error = 0.0;

    for(int i = 0; i < k; i++)
    {
        for(int j = 0; j < M; j++)
        {
            temp = y[i*M + j] - y2[i*M + j];
            error += temp*temp;
        }
    }

    return sqrt(error);
}

extern "C"
int main(int argc, char *argv[])
{
    
    
    /**************************************************************************/
    /* We first read in the matrix from a .mtx file.                          */
    /**************************************************************************/
    
    int ret_code;
    MM_typecode matcode;
    FILE *f;
    int M, N, nz;   
    int i, *I, *J, *row, *col;
    double *val, *valt;
    std::vector< std::pair< int, double > > rowval;

    if (argc < 2)
	{
		fprintf(stderr, "SpMV routine. Usage: %s [martix-market-filename]\n", 
		        argv[0]);
		exit(1);
	}
    else    
    { 
        if ((f = fopen(argv[1], "r")) == NULL) 
            exit(1);
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }


    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) && 
            mm_is_sparse(matcode) )
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz);
    if (ret_code !=0)   
        exit(1);


    /* reseve memory for matrices */

    I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));
    valt = (double *) malloc(nz * sizeof(double));   


    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d %lg\n", &I[i], &J[i], &valt[i]);
        I[i]--;  /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f !=stdin) fclose(f);
    
    /* convert the COO matrix to CSR */
    row = (int *) malloc((M+1) * sizeof(int));
    col = (int *) malloc(nz * sizeof(int));
    val = (double *) malloc(nz * sizeof(double));   
    
    // original code from  Nathan Bell and Michael Garland
    for ( i = 0; i < M; i++ )
        (row)[i] = 0;
    
    for ( i = 0; i < nz; i++ )
        (row)[I[i]]++;
    
    // cumulative sum the nnz per row to get row[]
    int cumsum;
    cumsum = 0;
    for( i = 0; i < M; i++ ) {
        int temp = row[i];
        (row)[i] = cumsum;
        cumsum += temp;
    }
    (row)[M] = nz;
    
    // write Aj,Ax into Bj,Bx
    for( i = 0; i < nz; i++ ) {
        int row_  = I[i];
        int dest = row[row_];
        col[dest] = J[i];
        val[dest] = valt[i];
        row[row_]++;
    }
    
    int last;
    last = 0;
    for( i = 0; i <= M; i++ ) {
        int temp  = (row)[i];
        (row)[i] = last;
        last      = temp;
    }
    
    (row)[M] = nz;

    // sort column indices within each row
    // copy into vector of pairs (column index, value), sort by column index
    for ( int k=0; k < M; ++k ) {
        int kk  = (row)[k];
        int len = (row)[k+1] - row[k];
        rowval.resize( len );
        for( i=0; i < len; ++i ) {
            rowval[i] = std::make_pair( col[kk+i], val[kk+i] );
        }
        std::sort( rowval.begin(), rowval.end(), compare_first );
        for( i=0; i < len; ++i ) {
            col[kk+i] = rowval[i].first;
            val[kk+i] = rowval[i].second;
        }
    }
    

    /**************************************************************************/
    /* Now we have the matrix in CPU main memory in CSR format.               */  
    /* The matrix A is stored as triplet A = [ row col val ].                 */
    /* We can now start with GPU allocation.                                  */
    /**************************************************************************/

    /* reserve memory for vectors x and y */
    int block_k;
    printf("What's the block size k?\n");
    scanf("%d", &block_k);
    int vecSize = block_k*N*sizeof(double);
    double *x, *y, *y2, *dx, *dy;
    x = (double*) malloc (vecSize);
    y = (double*) malloc (vecSize);
    y2 = (double*) malloc (vecSize);
    hipMalloc( (void**) &dx, vecSize);
    hipMalloc( (void**) &dy, vecSize);  
    
    /* reserve memory for the matrix */
    int *drow, *dcol;
    double *dval;
    
    hipMalloc( (void**) &dval, (nz*sizeof(double)));
    hipMalloc( (void**) &dcol, (nz*sizeof(int)));
    hipMalloc( (void**) &drow, ((M+1)*sizeof(int)));
    
    
    /* initialize vectors and send vectors and matrix to the GPU */
    srand(time(NULL));// init random number generator
    for (i=0; i < block_k*N; i++){
        //x[i] = (double) (rand()) / RAND_MAX;
        x[i] = 1;
        y[i] = 0.0;
        y2[i] = 0.0;
    }

    for( int i = 0; i < M; i++)
    {
        for (int j = 0; j < block_k; j++)
        {
            for(int l = row[i]; l < row[i+1]; l++)
            {
                y2[j*M + i] += val[l]*x[j*M + col[l]];
            }
        }
    }

    hipMemcpy( dx, x, vecSize, hipMemcpyHostToDevice );
    hipMemcpy( dy, y, vecSize, hipMemcpyHostToDevice );
    hipMemcpy( dval, val, (nz*sizeof(double)), hipMemcpyHostToDevice );
    hipMemcpy( dcol, col, (nz*sizeof(int)), hipMemcpyHostToDevice );
    hipMemcpy( drow, row, ((M+1)*sizeof(int)), hipMemcpyHostToDevice );
    
    
    /**************************************************************************/
    /* The setup for the CSR-SpMV kernel on GPU is ready. Now compute         */
    /* dy = dA* dx where dA is stored in [ drow dcol dval ].                  */  
    /**************************************************************************/   
     
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // here goes your kernel
    // define the kernel in a seperate file, please.
    // configure your compute grid using

    dim3 block(1024); 
    dim3 grid((int)(M*block_k/1024 + 1));

    hipEventRecord(start);

    spmv_kernel<<<grid,block>>>( M, block_k, N, nz, drow, dcol, dval, dx, dy);    

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // this is the runtime for 100 spmvs in ms
    printf("runtime [ms]: %f\n", milliseconds/( 100.0 ));
    
    
    /**************************************************************************/
    /* Copy the result back and check the correctness.                        */
    /**************************************************************************/ 
    
    hipMemcpy( y, dy, vecSize, hipMemcpyDeviceToHost);
    
    printf("Error norm: %lf \n", error_norm( M, block_k, y, y2));

    // How you check the correctness is up to you.
    // You can use MKL, use a CPU implementation of the SpMV, or any other 
    // validity test... 
    // You should compare the result in y with the reference result in y2
    // Explain what you do and how you do it.
    
    /**************************************************************************/
    /* In the end, we have to free all allocated memory.                      */
    /**************************************************************************/  
    
    hipFree(dx);
    hipFree(dy);
    hipFree(dval);
    hipFree(dcol);
    hipFree(drow);
    
    free(x);
    free(y);
    free(y2);
    free(val);
    free(col);
    free(row);
  
    free(valt);
    free(I);
    free(J);
    
	return 0;
}

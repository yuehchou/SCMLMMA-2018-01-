
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Kernel adding entries of the adjacent array entries (radius of 3) of a 1D array
//
// better approach
// * merge the 7 kernels into one

__global__ void kernel2(int n, int *a, int *b)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   
   if( i<n ){
     if(i>2)
        b[i]+=a[i-3];
     if(i>1)
        b[i]+=a[i-2];
     if(i>0)
        b[i]+=a[i-1];
     
     b[i]+=a[i]; 
     
     if(i<n-3)
        b[i]+=a[i+3];
     if(i<n-2)
        b[i]+=a[i+2];
     if(i<n-1)
        b[i]+=a[i+1];
   }
}


int main() {


  int n=2000000;
  int memSize = n*sizeof(int);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *a, *d_a;
  a = (int*) malloc (n*sizeof(*a));
  hipMalloc( (void**) &d_a, memSize);
  int *b, *d_b;
  b = (int*) malloc (n*sizeof(*b));
  hipMalloc( (void**) &d_b, memSize);
  
  for(int j=0; j<n; j++){
  	a[j] = j;
  	b[j] = 0;
  }

  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_b, b, memSize, hipMemcpyHostToDevice);

  dim3 block(256); 
  dim3 grid((n+block.x-1)/(block.x));
  
  hipEventRecord(start);
  kernel2<<<grid,block>>>(n,d_a,d_b);
  hipEventRecord(stop);

  hipMemcpy( b, d_b, memSize, hipMemcpyDeviceToHost);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("runtime [s]: %f\n", milliseconds/1000.0);
  
  for(int j=0; j<10; j++)
  	printf("%d\n",b[j]);
  	
  hipFree(d_a);
  free(a);
  hipFree(d_b);
  free(b);
  
  return 0;

}
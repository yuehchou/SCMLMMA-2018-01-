
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

/*
__global__ void kernel10(int *a)
{
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x); 
}

*/


__global__ void kernel10(int *a)
{
   if (threadIdx.x == 1 )
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x);
   if (threadIdx.x == 0 )
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x);
   if (threadIdx.x == 3 )
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x);
   if (threadIdx.x == 2 )
    printf("Hello from thread %d in block %d\n", threadIdx.x, blockIdx.x);

    
    
}



int main() {


  int n=20;
  int memSize = n*sizeof(int);

  int *a, *d_a;
  a = (int*) malloc (n*sizeof(*a));
  hipMalloc( (void**) &d_a, memSize);


  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);

  dim3 block(4); 
  dim3 grid(3);
  kernel10<<<grid,block>>>(d_a);

  hipMemcpy( a, d_a, memSize, hipMemcpyDeviceToHost);

  	
  hipFree(d_a);
  free(a);
  
  return 0;

}
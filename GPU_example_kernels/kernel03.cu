
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void kernel11(int *a, int *b, int *c)
{
   a[blockIdx.y*blockDim.x*gridDim.x+blockIdx.x*blockDim.x + threadIdx.x]=blockIdx.x;
   b[blockIdx.y*blockDim.x*gridDim.x+blockIdx.x*blockDim.x + threadIdx.x]=blockIdx.y;
   c[blockIdx.y*blockDim.x*gridDim.x+blockIdx.x*blockDim.x + threadIdx.x]=threadIdx.x;
}

int main() {


  int n=24;
  int memSize = n*sizeof(int);

  int *a, *b, *c, *d_a, *d_b, *d_c;
  a = (int*) malloc (n*sizeof(*a));
  b = (int*) malloc (n*sizeof(*b));
  c = (int*) malloc (n*sizeof(*c));
  hipMalloc( (void**) &d_a, memSize);
  hipMalloc( (void**) &d_b, memSize);
  hipMalloc( (void**) &d_c, memSize);


  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_b, b, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_c, c, memSize, hipMemcpyHostToDevice);

  int d1=4;
  int d2=2;
  int d3=1;
  int db=3;
  
  dim3 block(db); 
  dim3 grid(d1, d2, d3);
  kernel11<<<grid,block>>>(d_a, d_b, d_c);

  hipMemcpy( a, d_a, memSize, hipMemcpyDeviceToHost);
  hipMemcpy( b, d_b, memSize, hipMemcpyDeviceToHost);
  hipMemcpy( c, d_c, memSize, hipMemcpyDeviceToHost);
  
  for(int l=0; l<n; l++)
          printf("(%d, %d) -> %d\n",a[l], b[l], c[l]);
       
          printf("\n\n\n\n 2D output:\n\n");
          
 for(int k=0;k<d2; k++){
    for(int i=0; i<db; i++){
      for(int j=0; j<d1; j++){
        int l = j*db+i+k*db*d1;
        printf("(%d, %d) -> %d    ",a[l], b[l], c[l]);
      }
      printf("\n");
    }
    printf("\n");
  }
  
  hipFree(d_a);
  free(a);
  hipFree(d_b);
  free(b);
  hipFree(d_c);
  free(c);
  
  return 0;

}
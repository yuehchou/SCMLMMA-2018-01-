
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void kernel1(int *a)
{
   if(threadIdx.x > 2 )
        a[blockIdx.x*blockDim.x + threadIdx.x]=100;
    else
      a[blockIdx.x*blockDim.x + threadIdx.x]=blockIdx.x;  
}

int main() {


  int n=20;
  int memSize = n*sizeof(int);

  int *a, *d_a;
  a = (int*) malloc (n*sizeof(*a));
  hipMalloc( (void**) &d_a, memSize);


  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);

  dim3 block(4); 
  dim3 grid(n/block.x);
  kernel1<<<grid,block>>>(d_a);

  hipMemcpy( a, d_a, memSize, hipMemcpyDeviceToHost);

  for(int j=0; j<n; j++)
  	printf("%d\n",a[j]);
  	
  hipFree(d_a);
  free(a);
  
  return 0;

}
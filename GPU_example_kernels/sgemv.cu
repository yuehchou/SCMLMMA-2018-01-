
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


  __global__ void 
sgemv_rowmajor(int n, float a, float *m, float *x, float *y){

    int row = blockIdx.x*blockDim.x + threadIdx.x;
    float sum = 0.0;

    if (row < n){
        for( int col=0; col<n; col++){
            sum+= m[row*n+col] * x[col];
        }
        y[row] = a*sum;
    }  
}

  __global__ void 
sgemv_colmajor(int n, float a, float *m, float *x, float *y){

    int row = blockIdx.x*blockDim.x + threadIdx.x;
    float sum = 0.0;

    if (row < n){
        for( int col=0; col<n; col++){
            sum+= m[col*n+row] * x[col];
        }
        y[row] = a*sum;
    }  
}


int main() {


  int n=2000;
  int memSize = n*sizeof(int);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *a, *d_a;
  a = (float*) malloc (n*sizeof(*a));
  hipMalloc( (void**) &d_a, memSize);
  float *b, *d_b;
  b = (float*) malloc (n*sizeof(*b));
  hipMalloc( (void**) &d_b, memSize);
  float *m, *d_m;
  m = (float*) malloc (n*n*sizeof(*b));
  hipMalloc( (void**) &d_m, memSize*n);
  
  for(int j=0; j<n; j++){
  	a[j] = (float) j;
  	b[j] = (float) 0;
  	for(int k=0; k<n; k++)
  	    m[j*n+k] = (float) j+k;
  }
  
  float p = 1.0;

  hipMemcpy( d_a, a, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_b, b, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_m, m, memSize*n, hipMemcpyHostToDevice);

  dim3 block(256); 
  dim3 grid((n+block.x-1)/(block.x));
 
  hipEventRecord(start);
  sgemv_rowmajor<<<grid,block>>>(n, p, d_m, d_a, d_b);
  hipEventRecord(stop);

  hipMemcpy( b, d_b, memSize, hipMemcpyDeviceToHost);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("\n\nruntime row-major sgemv [s]: %f\n", milliseconds/1000.0);
  printf("\nresult:\n");
  for(int j=0; j<10; j++)
  	printf("%f\n",b[j]);
  	
  	
  hipEventRecord(start);
  sgemv_colmajor<<<grid,block>>>(n, p, d_m, d_a, d_b);
  hipEventRecord(stop);

  hipMemcpy( b, d_b, memSize, hipMemcpyDeviceToHost);

  hipEventElapsedTime(&milliseconds, start, stop);

  printf("\n\nruntime col-major sgemv[s]: %f\n", milliseconds/1000.0);
  printf("\nresult:\n");
  for(int j=0; j<10; j++)
  	printf("%f\n",b[j]);
  	
  hipFree(d_a);
  free(a);
  hipFree(d_b);
  free(b);
  hipFree(d_m);
  free(m);
  
  return 0;

}